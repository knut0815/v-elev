#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>
#include <iterator>

#include "renderer.h"
#include "voxel_model.h"
#include ""
#include "pdf.h"
#include "material.h"

void err(hipError_t err, char *msg)
{
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to %s (error code %s)!\n", msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

struct pixel_compare {
	const pixel* pixels;
	const uint ns;
	pixel_compare(const pixel* _pixels, uint _ns): pixels(_pixels), ns(_ns) {}

	bool operator() (int p0, int p1) {
		return pixels[p0].done < pixels[p1].done;
	}
};


void renderer::prepare_kernel() {
	const unsigned int num_pixels = nx*ny;
	const uint unit_numpixels = num_pixels / num_units; //TODO make sure we don't miss any rays because of precision loss

	remaining_pixels = num_pixels;
	next_pixel = 0;
	total_rays = 0;

	// allocate device memory for input
    d_heightmap = NULL;
	err(hipMalloc((void **)&d_heightmap, model->size.x*model->size.z*sizeof(unsigned char)), "allocate device d_scene");

	// Copy the host input in host memory to the device input in device memory
	err(hipMemcpy(d_heightmap, model->heightmap,model->size.x*model->size.z*sizeof(unsigned char), hipMemcpyHostToDevice), "copy scene from host to device");

	wunits = new work_unit*[num_units];
	uint cur_idx = 0;
	for (uint unit = 0; unit < num_units; unit++) {
		uint next_idx = cur_idx + unit_numpixels;
		work_unit *wu = new work_unit(cur_idx, next_idx);
		const uint unit_len = wu->length();

		wu->pixel_idx = new int[unit_len];
		wu->samples = new sample[unit_len];

		wu->pixels = new pixel[unit_len];
		for (uint i = 0; i < unit_len; i++)
			wu->pixels[i].samples = 1;

		wu->h_colors = new float3[unit_len];
		for (uint i = 0; i < unit_len; i++)
			wu->h_colors[i] = make_float3(0);

		err(hipHostMalloc(&wu->h_rays, unit_len * sizeof(ray)), "allocate h_rays");
		err(hipMalloc((void **)&(wu->d_rays), unit_len * sizeof(ray)), "allocate device d_rays");
		err(hipMalloc((void **)&(wu->d_hits), unit_len * sizeof(cu_hit)), "allocate device d_hits");
		err(hipHostMalloc(&(wu->h_clrs), unit_len * sizeof(clr_rec)), "allocate h_clrs");
		err(hipMalloc((void **)&(wu->d_clrs), unit_len * sizeof(clr_rec)), "allocate device d_clrs");
		err(hipStreamCreate(&wu->stream), "cuda stream create");

		wunits[unit] = wu;
		cur_idx = next_idx;
	}

	generate_rays();
}

void renderer::update_camera()
{
	const unsigned int num_pixels = numpixels();

	for (uint unit = 0; unit < num_units; unit++) {
		work_unit* wu = wunits[unit];
		wu->done = false;
		for (uint i = 0; i < wu->length(); i++) {
			wu->pixels[i].samples = 1;
			wu->pixels[i].done = 0;
			wu->h_colors[i] = make_float3(0, 0, 0);
		}
	}

	generate_rays();
	num_runs = 0;
}

void renderer::generate_rays() {
	uint ray_idx = 0;
	for (int j = ny - 1; j >= 0; j--)
		for (int i = 0; i < nx; ++i, ++ray_idx) {
			// for initial generation ray_idx == pixelId
			const uint unit_idx = get_unitIdx(ray_idx);
			generate_ray(wunits[unit_idx], ray_idx, i, j);
		}
}

inline void renderer::generate_ray(work_unit* wu, const uint sampleId, int x, int y) {
	// even though we can compute pixelId from (x,y), we still need the sampleId as its not necessarely the same (as more than a single sample point to the same pixel)
	const float u = float(x + drand48()) / float(nx);
	const float v = float(y + drand48()) / float(ny);
	const uint local_ray_idx = sampleId - wu->start_idx;
	cam->get_ray(u, v,wu->h_rays[local_ray_idx]);
	wu->samples[local_ray_idx] = sample(get_pixelId(x, y));
}

__global__ void hit_scene(const ray* rays, const uint num_rays, const unsigned char* heightmap, const uint3 model_size, float t_min, float t_max, cu_hit* hits)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i >= num_rays) return;

	const ray *r = &(rays[i]);
	const voxelModel model(heightmap, model_size);
	cu_hit hit;
	if (!model.hit(*r, t_min, t_max, hit)) {
		hits[i].hit_face = NO_HIT;
		return;
	}

	hits[i].hit_face = hit.hit_face;
	hits[i].hit_t = hit.hit_t;
}

__global__ void simple_color(const ray* rays, const uint num_rays, const cu_hit* hits, clr_rec* clrs, const uint seed, const float3 albedo, const int max_depth) {
	const int ray_idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (ray_idx >= num_rays) return;

	const ray& r = rays[ray_idx];
	const cu_hit hit(hits[ray_idx]);
	clr_rec& crec = clrs[ray_idx];

	if (hit.hit_face == NO_HIT) {
		// no intersection with spheres, return sky color
		float3 unit_direction = normalize(r.direction);
		float t = 0.5*(unit_direction.y + 1.0);
		crec.color = 1.0* ((1 - t)*make_float3(1.0, 1.0, 1.0) + t*make_float3(0.5, 0.7, 1.0));
		crec.done = true;
		return;
	}

	const float3 hit_n = make_float3(
		-1 * (hit.hit_face == X)*signum(r.direction.x),
		-1 * (hit.hit_face == Y)*signum(r.direction.y),
		-1 * (hit.hit_face == Z)*signum(r.direction.z)
	);

	hit_record rec(r.point_at_parameter(hit.hit_t), hit_n);
	hiprandStatePhilox4_32_10_t localState;
	hiprand_init(0, seed*blockDim.x + threadIdx.x, 0, &localState);
	lambertian mat(albedo);

	scatter_record srec;
	mat.scatter(rec, srec);
	srec.scattered = ray(rec.hit_p, srec.pdf_ptr->generate(&localState));
	const float pdf_val = srec.pdf_ptr->value(srec.scattered.direction);
	if (pdf_val > 0) {
		const float scattering_pdf = mat.scattering_pdf(rec, srec.scattered);
		srec.attenuation *= scattering_pdf / pdf_val;

		crec.origin = srec.scattered.origin;
		crec.direction = srec.scattered.direction;
		crec.color = srec.attenuation;
		crec.done = false;

		// following code can be useful to debug rendering issues
		//const uint max_dir = max_id(srec.scattered.direction);
		//crec.color = (make_float3(
		//	(max_dir == 0)*signum(srec.scattered.direction.x),
		//	(max_dir == 1)*signum(srec.scattered.direction.y),
		//	(max_dir == 2)*signum(srec.scattered.direction.z)
		//) + 1) / 2;
		//crec.color = (normalize(hit_n) + 1) / 2;
		//crec.done = true;
	} else {
		crec.color = make_float3(0, 0, 0);
		crec.done = true;
	}
	delete srec.pdf_ptr;
}

void renderer::copy_rays_to_gpu(const work_unit* wu) {
	err(hipMemcpyAsync(wu->d_rays, wu->h_rays, wu->length() * sizeof(ray), hipMemcpyHostToDevice, wu->stream), "copy rays from host to device");
}

void renderer::copy_colors_from_gpu(const work_unit* wu) {
	err(hipMemcpyAsync(wu->h_clrs, wu->d_clrs, wu->length() * sizeof(clr_rec), hipMemcpyDeviceToHost, wu->stream), "copy results from device to host");
}

void renderer::start_kernel(const work_unit* wu) {
	int threadsPerBlock = 128;
	int blocksPerGrid = (wu->length() + threadsPerBlock - 1) / threadsPerBlock;
	hit_scene <<<blocksPerGrid, threadsPerBlock, 0, wu->stream >>>(wu->d_rays, wu->length(), d_heightmap, model->size, 0.1f, FLT_MAX, wu->d_hits);
	simple_color <<<blocksPerGrid, threadsPerBlock, 0, wu->stream >>>(wu->d_rays, wu->length(), wu->d_hits, wu->d_clrs, num_runs++, model_albedo, max_depth);
}

void renderer::render_work_unit(uint unit_idx) {
	work_unit* wu = wunits[unit_idx];
	while (!wu->done) {
		wu->num_iter++;
		clock_t start = clock();
		copy_rays_to_gpu(wu);
		start_kernel(wu);
		copy_colors_from_gpu(wu);
		hipStreamQuery(wu->stream); // flush stream to start the kernel 
		hipStreamSynchronize(wu->stream);
		clock_t end = clock();
		wu->gpu_time += end - start;
		compact_rays(wu);
		wu->cpu_time += clock() - end;
	}
}

void renderer::compact_rays(work_unit* wu) {
	uint done_samples = 0;
	bool not_done = false;
	for (uint i = 0; i < wu->length(); ++i) {
		const clr_rec& crec = wu->h_clrs[i];
		sample& s = wu->samples[i];
		const uint local_pixelId = s.pixelId - wu->start_idx;
		s.done = crec.done || s.depth == max_depth;
		if (s.done) {
			if (crec.done) wu->h_colors[local_pixelId] += s.not_absorbed*crec.color;
			++(wu->pixels[local_pixelId].done);
			++done_samples;
		} else {
			s.not_absorbed *= crec.color;
			wu->h_rays[i].origin = crec.origin;
			wu->h_rays[i].direction = crec.direction;
			++s.depth;
		}
		not_done = not_done || (wu->pixels[local_pixelId].done < ns);
	}

	if (done_samples > 0 && not_done) {
		// sort uint ray [wu->start_idx, wu->end_idx[
		for (uint i = 0; i < wu->length(); ++i) wu->pixel_idx[i] = i;
		std::sort(wu->pixel_idx, wu->pixel_idx + wu->length(), pixel_compare(wu->pixels, ns));
		uint sampled = 0;
		for (uint i = 0; i < wu->length(); ++i) {
			const uint sId = wu->start_idx + i;
			sample& s = wu->samples[i];
			if (s.done) {
				// generate new ray
				const uint local_pixelId = wu->pixel_idx[sampled++];
				const uint pixelId = wu->start_idx + local_pixelId;
				wu->pixels[local_pixelId].samples++;
				// then, generate a new sample
				const unsigned int x = pixelId % nx;
				const unsigned int y = ny - 1 - (pixelId / nx);
				generate_ray(wu, sId, x, y);
			}
		}
	}

	wu->done = !not_done;
}

void renderer::destroy() {
	// Free device global memory
	err(hipFree(d_heightmap), "free device d_heightmap");

	for (uint unit = 0; unit < num_units; unit++) {
		work_unit *wu = wunits[unit];
		err(hipFree(wu->d_rays), "free device d_rays");
		err(hipFree(wu->d_hits), "free device d_hits");
		err(hipFree(wu->d_clrs), "free device d_clrs");

		err(hipStreamDestroy(wu->stream), "destroy cuda stream");

		hipHostFree(wu->h_clrs);
		hipHostFree(wu->h_rays);

		delete[] wu->pixel_idx;
		delete[] wu->samples;
		delete[] wu->pixels;
		delete[] wu->h_colors;
	}

	// Free host memory
	delete[] wunits;
}